#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void computeCoordsJulia(double* coordX, double* coordY, int width, int height, double minX, double minY, double rangeX, double rangeY)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int y = i / width;
    int x = i % width;
    coordX[i] = minX + (rangeX / width) * x;
    coordY[i] = minY + (rangeY / height) * y;
}

__global__ void findIterationsJulia(int* iterArray, double* coordX, double* coordY, double customX, double customY)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    double realZ = coordX[i] + customX;
    double imagZ = coordY[i] + customY;
    double realZ2 = 0;
    double imagZ2 = 0;
    while (iterArray[i] < 255)
    {
        iterArray[i]++;
        realZ2 = realZ * realZ;
        imagZ2 = imagZ * imagZ;
        if (realZ2 + imagZ2 > 4)
        {
            break;
        }
        imagZ = 2 * realZ * imagZ + customY;
        realZ = realZ2 - imagZ2 + customX;
    }
}

// Helper function for using CUDA to add vectors in parallel.
extern "C" __declspec(dllexport) void computeJulia(int* iterArray, int width, int height, double minX, double minY, double rangeX, double rangeY, double customX, double customY)
{
    hipError_t cudaStatus;
    double* coordX;
    double* coordY;
    int* iterArrayGpu;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    hipMalloc(&coordX, width * height * sizeof(double));
    hipMalloc(&coordY, width * height * sizeof(double));
    hipMalloc(&iterArrayGpu, width * height * sizeof(int));


    // Launch a kernel on the GPU with one thread for each element.
    computeCoordsJulia << <(width * height) / 256, 256 >> > (coordX, coordY, width, height, minX, minY, rangeX, rangeY);
    findIterationsJulia << <(width * height) / 256, 256 >> > (iterArrayGpu, coordX, coordY, customX, customY);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(iterArray, iterArrayGpu, width * height * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(coordX);
    hipFree(coordY);
    hipFree(iterArrayGpu);
}
